#include "hip/hip_runtime.h"
/*****************************************************************/
/*	             THIS FILE WILL BE GENERATED                     */
/*	                      DO NOT EDIT!                           */
/*****************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "converter.cu"
#include "writeRecord.cu"

#define RECORDLENGTH 120
struct recordData
{
	// 03 RECID        PIC 9(05) COMP-3 VALUE ZERO. // 3
	uint32_t 	RECID;
	// 03 SYSTEMID     PIC X(04) VALUE 'S085'. // 4
	char		*SYSTEMID[4];
	// 03 MANDID       PIC 9(05) COMP-3 VALUE 10010. // 3
	uint32_t	MANDID;
	// 03 NAME         PIC X(40) VALUE 'MAX MUSTER'. // 40
	char 		*NAME[40];
	// 03 POLNR        PIC 9(11) COMP-3 VALUE 0100001. // 6
	uint32_t 	POLNR;
	// 03 RISPA        PIC 9(03) COMP-3 VALUE 207. // 2
	uint32_t 	RISPA;
	// 03 WAEHR        PIC X(03) VALUE 'EUR'. // 3
	char 		*WAEHR[3];
	// 03 PRAEMIE      PIC S9(9)V99 COMP-3 VALUE 228.30. // 5
	int32_t		PRAEMIE;
	// 03 INFO         PIC X(50) VALUE 'ICH BIN EIN SATZ.'. // 50
	char 		*INFO[50];
	// 03 ENDE         PIC X(03) VALUE LOW-VALUE. // 3
	char 		*ENDE[3];
};

__device__ int getGlobalIdx_1D_1D()
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void decodeRecord ( int numOfCores, int recordNum, uint8_t *inputMemAddress , uint8_t *outputMemAddress ) {
	int recordAddress;
	int threadIdx;
	threadIdx = getGlobalIdx_1D_1D();
	recordAddress = (threadIdx * RECORDLENGTH);
	outputMemAddress[threadIdx] = recordAddress; // recordAddress; // inputMemAddress[ ( uint8_t ) recordAddress ];

	struct recordData theRecord;
	comp3ToInt ( recordAddress, 5, &theRecord.RECID );
		recordAddress = recordAddress + 5; 
	charToCharArray ( recordAddress, 4, *theRecord.SYSTEMID );
		recordAddress = recordAddress + 4; 
	comp3ToInt ( recordAddress, 5, &theRecord.MANDID );
		recordAddress = recordAddress + 5; 
	charToCharArray ( recordAddress, 40, *theRecord.NAME );
		recordAddress = recordAddress + 40; 
	comp3ToInt ( recordAddress, 11, &theRecord.POLNR );
		recordAddress = recordAddress + 11; 
	comp3ToInt ( recordAddress, 3, &theRecord.RISPA );
		recordAddress = recordAddress + 3; 
	charToCharArray ( recordAddress, 3, *theRecord.WAEHR );
		recordAddress = recordAddress + 3; 
	comp3ToSignedInt ( recordAddress, 9, &theRecord.PRAEMIE );
		recordAddress = recordAddress + 9; 
	charToCharArray ( recordAddress, 50, *theRecord.INFO );
		recordAddress = recordAddress + 50; 
	charToCharArray ( recordAddress, 3, *theRecord.ENDE );
		recordAddress = recordAddress + 3; 

}
