#include "hip/hip_runtime.h"
/*****************************************************************/
/*               THIS FILE WILL BE GENERATED                     */
/*                        DO NOT EDIT!                           */
/*****************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "converter.cu"
#include "writeRecord.cu"

#define RECORDLENGTH 120

__device__ int getGlobalIdx_1D_1D() {
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__device__ int getGlobalIdx_1D_2D() {
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void decodeRecord ( int numOfCores, int recordNum, uint8_t *inputMemAddress , uint8_t *outputMemAddress ) {
	int recordAddress,outputAddress;
	int threadIdx;
	threadIdx = getGlobalIdx_1D_2D(); // getGlobalIdx_1D_1D();
	recordAddress = (threadIdx * RECORDLENGTH);
	outputAddress = (threadIdx * RECORDLENGTH);
	outputMemAddress[threadIdx] = inputMemAddress[ recordAddress ]; // recordAddress; // inputMemAddress[ ( uint8_t ) recordAddress ];
	int bcdIntegerLength = 5;
	// 03 RECID        PIC 9(05) COMP-3 VALUE ZERO. // 3
	comp3ToInt ( inputMemAddress, recordAddress, bcdIntegerLength, 3, 4, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 3; 
		outputAddress = outputAddress + 4;
	// 03 SYSTEMID     PIC X(04) VALUE 'S085'. // 4
	charToCharArray ( inputMemAddress, recordAddress, 4, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 4; 
		outputAddress = outputAddress + 4;
	// 03 MANDID       PIC 9(05) COMP-3 VALUE 10010. // 3
	comp3ToInt ( inputMemAddress, recordAddress, bcdIntegerLength, 3, 4, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 3; 
		outputAddress = outputAddress + 4;
	// 03 NAME         PIC X(40) VALUE 'MAX MUSTER'. // 40
	charToCharArray ( inputMemAddress, recordAddress, 40, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 40; 
		outputAddress = outputAddress + 40;
	// 03 POLNR        PIC 9(11) COMP-3 VALUE 0100001. // 6
	bcdIntegerLength = 11;
	comp3ToInt ( inputMemAddress, recordAddress, bcdIntegerLength, 6, 4, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 6; 
		outputAddress = outputAddress + 4;
	// 03 RISPA        PIC 9(03) COMP-3 VALUE 207. // 2
	bcdIntegerLength = 3;
	comp3ToInt ( inputMemAddress, recordAddress, bcdIntegerLength, 2, 4, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 2; 
		outputAddress = outputAddress + 4;
	// 03 WAEHR        PIC X(03) VALUE 'EUR'. // 3
	charToCharArray ( inputMemAddress, recordAddress, 3, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 3; 
		outputAddress = outputAddress + 3;
	// 03 PRAEMIE      PIC S9(9)V99 COMP-3 VALUE 228.30. // 5
	bcdIntegerLength = 9;
	comp3ToInt ( inputMemAddress, recordAddress, bcdIntegerLength, 6, 4, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 6; 
		outputAddress = outputAddress + 4;
	// 03 INFO         PIC X(50) VALUE 'ICH BIN EIN SATZ.'. // 50
	charToCharArray ( inputMemAddress, recordAddress, 50, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 50; 
		outputAddress = outputAddress + 50;
	// 03 ENDE         PIC X(03) VALUE LOW-VALUE. // 3
	charToCharArray ( inputMemAddress, recordAddress, 3, outputMemAddress, outputAddress );
	//	recordAddress = recordAddress + 3; 
	//	outputAddress = outputAddress + 3;
}
