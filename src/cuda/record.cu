#include "hip/hip_runtime.h"

/*****************************************************************/
/*               THIS FILE WILL BE GENERATED                     */
/*                        DO NOT EDIT!                           */
/*****************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

    
#define NUMOFRECORDS 10000
#define RECORDLENGTH 64
#define OUTPUTRECORDLENGTH 131 
#define BINARYSIZE RECORDLENGTH * NUMOFRECORDS 
#define OUTPUTBINARYSIZE OUTPUTRECORDLENGTH * NUMOFRECORDS
#define NUMOFCORES 1024

#define BLOCKSPERGRID 1024

#define DELIMITER 0x3b
#define COMMA 0x2c

#include "converter.cu"

__device__ int getGlobalIdx_1D_1D() {
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__device__ int getGlobalIdx_1D_2D() {
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void decodeRecord ( uint8_t *inputMemAddress , uint8_t *outputMemAddress ) {
	int recordAddress,outputAddress;
	int threadIdx;
	int bcdIntegerLength = 0;
	threadIdx = getGlobalIdx_1D_2D(); // getGlobalIdx_1D_1D();
	if (threadIdx < NUMOFRECORDS ) {
		recordAddress = (threadIdx * RECORDLENGTH);
		outputAddress = (threadIdx * OUTPUTRECORDLENGTH);

    		// DECIMAL: POLNR
		bcdIntegerLength = 11;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 6, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 6;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: SPARTE
		bcdIntegerLength = 1;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 1, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 1;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: TARIF
		bcdIntegerLength = 5;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 3, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 3;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: RISIKO
		bcdIntegerLength = 3;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 2, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 2;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: SCHICHT
		bcdIntegerLength = 5;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 3, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 3;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;

         
      		// DECIMAL: AEART
		bcdIntegerLength = 3;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 2, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 2;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// INTEGER: BONUS
		compToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 2, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 4;
			outputAddress = outputAddress + 5;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: KFZLFDNR
		bcdIntegerLength = 3;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 2, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 2;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;

         
      
         
      
         
      
         
      
         
      		// INTEGER: STATZAHL
		compToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 2, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 4;
			outputAddress = outputAddress + 5;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// CHAR: KZGRUENK
		charToCharArray ( inputMemAddress, recordAddress, 1, outputMemAddress, outputAddress);
			recordAddress = recordAddress + 1;
			outputAddress = outputAddress + 2;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: AUTTAR
		bcdIntegerLength = 1;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 1, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 1;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: BASISPREX
		bcdIntegerLength = 13;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 7, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 7;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: BASISPRIN
		bcdIntegerLength = 13;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 7, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 7;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: BMSTUFE
		bcdIntegerLength = 5;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 3, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 3;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: BMJAHR
		bcdIntegerLength = 7;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 4, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 4;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: BLOCKNR
		bcdIntegerLength = 3;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 2, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 2;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;

			outputMemAddress[outputAddress] = 0x0d;
			outputMemAddress[outputAddress + 1] = 0x0a;
	}

}
      
    