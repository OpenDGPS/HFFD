#include "hip/hip_runtime.h"
/*****************************************************************/
/*               THIS FILE WILL BE GENERATED                     */
/*                        DO NOT EDIT!                           */
/*****************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "converter.cu"
#include "writeRecord.cu"

#define NUMOFRECORDS 13000
#define RECORDLENGTH 120
#define OUTPUTRECORDLENGTH 150

__device__ int getGlobalIdx_1D_1D() {
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__device__ int getGlobalIdx_1D_2D() {
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void decodeRecord ( int numOfCores, int recordNum, uint8_t *inputMemAddress , uint8_t *outputMemAddress ) {
	int recordAddress,outputAddress;
	int threadIdx;
	threadIdx = getGlobalIdx_1D_2D(); // getGlobalIdx_1D_1D();
	recordAddress = (threadIdx * RECORDLENGTH);
	outputAddress = (threadIdx * OUTPUTRECORDLENGTH);
	outputMemAddress[threadIdx] = inputMemAddress[ recordAddress ]; // recordAddress; // inputMemAddress[ ( uint8_t ) recordAddress ];
	int bcdIntegerLength = 5;
	// 03 RECID        PIC 9(05) COMP-3 VALUE ZERO. // 3
	// InLength total: 3 4 3 40 6 2 3 6 50 3
	// Length total: 8 4 8 40 8 8 3 8 50 3
	comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 3, 8, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 3; 
		outputAddress = outputAddress + 8;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 SYSTEMID     PIC X(04) VALUE 'S085'. // 4
	charToCharArray ( inputMemAddress, recordAddress, 4, outputMemAddress, outputAddress );
	// outputMemAddress[outputAddress] = threadIdx;
		recordAddress = recordAddress + 4; 
		outputAddress = outputAddress + 5;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 MANDID       PIC 9(05) COMP-3 VALUE 10010. // 3
	comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 3, 8, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 3; 
		outputAddress = outputAddress + 9;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 NAME         PIC X(40) VALUE 'MAX MUSTER'. // 40
	charToCharArray ( inputMemAddress, recordAddress, 40, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 40; 
		outputAddress = outputAddress + 41;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 POLNR        PIC 9(11) COMP-3 VALUE 0100001. // 6
	bcdIntegerLength = 11;
	comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 6, 8, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 6; 
		outputAddress = outputAddress + 9;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 RISPA        PIC 9(03) COMP-3 VALUE 207. // 2
	bcdIntegerLength = 3;
	comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 2, 8, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 2; 
		outputAddress = outputAddress + 9;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 WAEHR        PIC X(03) VALUE 'EUR'. // 3
	charToCharArray ( inputMemAddress, recordAddress, 3, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 3; 
		outputAddress = outputAddress + 4;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 PRAEMIE      PIC S9(9)V99 COMP-3 VALUE 228.30. // 5
	bcdIntegerLength = 11;
	comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 6, 8, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 6; 
		outputAddress = outputAddress + 9;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 INFO         PIC X(50) VALUE 'ICH BIN EIN SATZ.'. // 50
	charToCharArray ( inputMemAddress, recordAddress, 50, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 50; 
		outputAddress = outputAddress + 51;
		outputMemAddress[outputAddress - 1] = DELIMITER;
	// 03 ENDE         PIC X(03) VALUE LOW-VALUE. // 3
	charToCharArray ( inputMemAddress, recordAddress, 3, outputMemAddress, outputAddress );
		recordAddress = recordAddress + 3; 
		outputAddress = outputAddress + 3;
		outputMemAddress[outputAddress] = 0x0d;
		outputMemAddress[outputAddress + 1] = 0x0a;
}
