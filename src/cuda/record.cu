#include "hip/hip_runtime.h"
/*****************************************************************/
/*               THIS FILE WILL BE GENERATED                     */
/*                        DO NOT EDIT!                           */
/*****************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "converter.cu"
#include "writeRecord.cu"

#define RECORDLENGTH 120
typedef struct recordData
{
	// 03 RECID        PIC 9(05) COMP-3 VALUE ZERO. // 3
	uint32_t 	RECID;
	// 03 SYSTEMID     PIC X(04) VALUE 'S085'. // 4
	uint32_t	SYSTEMID_0;
	uint32_t	SYSTEMID_1;
	uint32_t	SYSTEMID_2;
	uint32_t	SYSTEMID_3;
	// 03 MANDID       PIC 9(05) COMP-3 VALUE 10010. // 3
	uint32_t	MANDID;
	// 03 NAME         PIC X(40) VALUE 'MAX MUSTER'. // 40
	char 		*NAME[40];
	// 03 POLNR        PIC 9(11) COMP-3 VALUE 0100001. // 6
	uint32_t 	POLNR;
	// 03 RISPA        PIC 9(03) COMP-3 VALUE 207. // 2
	uint32_t 	RISPA;
	// 03 WAEHR        PIC X(03) VALUE 'EUR'. // 3
	char 		*WAEHR[3];
	// 03 PRAEMIE      PIC S9(9)V99 COMP-3 VALUE 228.30. // 5
	int32_t		PRAEMIE;
	// 03 INFO         PIC X(50) VALUE 'ICH BIN EIN SATZ.'. // 50
	char 		*INFO[50];
	// 03 ENDE         PIC X(03) VALUE LOW-VALUE. // 3
	char 		*ENDE[3];
} theRecords;

// int outputLength[10] = {4,4,4,40,4,4,3,4,50,3};

__device__ int getGlobalIdx_1D_1D()
{
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__global__ void decodeRecord ( int numOfCores, int recordNum, uint8_t *inputMemAddress , uint8_t *outputMemAddress ) {
	int recordAddress,outputAddress;
	int threadIdx;
	threadIdx = getGlobalIdx_1D_1D();
	recordAddress = (threadIdx * RECORDLENGTH);
	outputAddress = (threadIdx * RECORDLENGTH);
	outputMemAddress[threadIdx] = inputMemAddress[ recordAddress ]; // recordAddress; // inputMemAddress[ ( uint8_t ) recordAddress ];
	theRecords theRecord;
	theRecords * thisRecord = &theRecord;
	// recordData theRecords;
	// theRecords *theRecord = malloc(sizeof(theRecords));
	thisRecord->RECID = 14;
	comp3ToInt ( inputMemAddress, recordAddress, 3, &thisRecord->RECID );
		outputMemAddress[outputAddress] = thisRecord->RECID >> 24;
		outputMemAddress[outputAddress + 1] = thisRecord->RECID >> 16;
		outputMemAddress[outputAddress + 2] = thisRecord->RECID >> 8;
		outputMemAddress[outputAddress + 3] = thisRecord->RECID >> 0;
		recordAddress = recordAddress + 3; 
		outputAddress = outputAddress + 4;
	charToCharArray ( inputMemAddress, recordAddress, 1, &thisRecord->SYSTEMID_0 );
		outputMemAddress[outputAddress] = thisRecord->SYSTEMID_0;
		recordAddress = recordAddress + 1; 
		outputAddress = outputAddress + 1;
	charToCharArray ( inputMemAddress, recordAddress, 1, &thisRecord->SYSTEMID_1 );
		outputMemAddress[outputAddress] = thisRecord->SYSTEMID_1;
		recordAddress = recordAddress + 1; 
		outputAddress = outputAddress + 1;
	charToCharArray ( inputMemAddress, recordAddress, 1, &thisRecord->SYSTEMID_2 );
		outputMemAddress[outputAddress] = thisRecord->SYSTEMID_2;
		recordAddress = recordAddress + 1; 
		outputAddress = outputAddress + 1;
	charToCharArray ( inputMemAddress, recordAddress, 1, &thisRecord->SYSTEMID_3 );
		outputMemAddress[outputAddress] = thisRecord->SYSTEMID_3;
		recordAddress = recordAddress + 1; 
		outputAddress = outputAddress + 1;
/*	comp3ToInt ( inputMemAddress, recordAddress, 5, &theRecord.MANDID );
		recordAddress = recordAddress + 5; 
	charToCharArray ( recordAddress, 40, *theRecord.NAME );
		recordAddress = recordAddress + 40; 
	comp3ToInt ( inputMemAddress, recordAddress, 11, &theRecord.POLNR );
		recordAddress = recordAddress + 11; 
	comp3ToInt ( inputMemAddress, recordAddress, 3, &theRecord.RISPA );
		recordAddress = recordAddress + 3; 
	charToCharArray ( recordAddress, 3, *theRecord.WAEHR );
		recordAddress = recordAddress + 3; 
	comp3ToSignedInt ( recordAddress, 9, &theRecord.PRAEMIE );
		recordAddress = recordAddress + 9; 
	charToCharArray ( recordAddress, 50, *theRecord.INFO );
		recordAddress = recordAddress + 50; 
	charToCharArray ( recordAddress, 3, *theRecord.ENDE );
		recordAddress = recordAddress + 3; */
}
