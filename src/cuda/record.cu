#include "hip/hip_runtime.h"

/*****************************************************************/
/*               THIS FILE WILL BE GENERATED                     */
/*                        DO NOT EDIT!                           */
/*****************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

    
#define NUMOFRECORDS 223
#define RECORDLENGTH 63
#define OUTPUTRECORDLENGTH 81 
#define BINARYSIZE RECORDLENGTH * NUMOFRECORDS 
#define OUTPUTBINARYSIZE OUTPUTRECORDLENGTH * NUMOFRECORDS
#define NUMOFCORES 1024

#define BLOCKSPERGRID 1024

#define DELIMITER 0x3b
#define COMMA 0x2c

#include "converter.cu"

__device__ int getGlobalIdx_1D_1D() {
	return blockIdx.x *blockDim.x + threadIdx.x;
}

__device__ int getGlobalIdx_1D_2D() {
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void decodeRecord ( uint8_t *inputMemAddress , uint8_t *outputMemAddress ) {
	int recordAddress,outputAddress;
	int threadIdx;
	int bcdIntegerLength = 0;
	threadIdx = getGlobalIdx_1D_2D(); // getGlobalIdx_1D_1D();
	if (threadIdx < NUMOFRECORDS ) {
		recordAddress = (threadIdx * RECORDLENGTH);
		outputAddress = (threadIdx * OUTPUTRECORDLENGTH);

    		// DECIMAL: BUENDELPOLNR
		bcdIntegerLength = 11;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 6, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 6;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: VERSION
		bcdIntegerLength = 5;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 3, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 3;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: SPARTENPOLNR
		bcdIntegerLength = 11;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 6, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 6;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// CHAR: PRODUKT
		charToCharArray ( inputMemAddress, recordAddress, 5, outputMemAddress, outputAddress);
			recordAddress = recordAddress + 5;
			outputAddress = outputAddress + 6;
			outputMemAddress[outputAddress - 1] = DELIMITER;

         
      		// DECIMAL: VB
		bcdIntegerLength = 9;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 5, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 5;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// CHAR: KZVALID
		charToCharArray ( inputMemAddress, recordAddress, 1, outputMemAddress, outputAddress);
			recordAddress = recordAddress + 1;
			outputAddress = outputAddress + 2;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: BETREUER
		bcdIntegerLength = 7;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 4, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 4;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: ANTRAGSART
		bcdIntegerLength = 1;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 1, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 1;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// CHAR: TC_KZ
		charToCharArray ( inputMemAddress, recordAddress, 3, outputMemAddress, outputAddress);
			recordAddress = recordAddress + 3;
			outputAddress = outputAddress + 4;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// DECIMAL: SPARTE
		bcdIntegerLength = 1;
		comp3ToIntSerial ( inputMemAddress, recordAddress, bcdIntegerLength, 1, 8, outputMemAddress, outputAddress, 0 );
			recordAddress = recordAddress + 1;
			outputAddress = outputAddress + 9;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// CHAR: KZDUPO
		charToCharArray ( inputMemAddress, recordAddress, 1, outputMemAddress, outputAddress);
			recordAddress = recordAddress + 1;
			outputAddress = outputAddress + 2;
			outputMemAddress[outputAddress - 1] = DELIMITER;
		// CHAR: VERARB_STAT
		charToCharArray ( inputMemAddress, recordAddress, 1, outputMemAddress, outputAddress);
			recordAddress = recordAddress + 1;
			outputAddress = outputAddress + 2;
			outputMemAddress[outputAddress - 1] = DELIMITER;

			outputMemAddress[outputAddress] = 0x0d;
			outputMemAddress[outputAddress + 1] = 0x0a;
	}

}
      
    