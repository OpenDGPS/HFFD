#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

struct recordData
{
	// 03 RECID        PIC 9(05) COMP-3 VALUE ZERO.
	uint32_t 	RECID;
	// 03 SYSTEMID     PIC X(04) VALUE 'S085'.
	char		SYSTEMID[4];
	// 03 MANDID       PIC 9(05) COMP-3 VALUE 10010.
	uint32_t	MANDID;
	// 03 NAME         PIC X(40) VALUE 'MAX MUSTER'.
	char 		NAME[40];
	// 03 POLNR        PIC 9(11) COMP-3 VALUE 0100001.
	uint32_t 	POLNR;
	// 03 RISPA        PIC 9(03) COMP-3 VALUE 207.
	uint32_t 	RISPA;
	// 03 WAEHR        PIC X(03) VALUE 'EUR'.
	char 		WAEHR[3];
	// 03 PRAEMIE      PIC S9(9)V99 COMP-3 VALUE 228.30.
	int32_t		PRAEMIE;
	// 03 INFO         PIC X(50) VALUE 'ICH BIN EIN SATZ.'.
	char 		INFO[50];
	// 03 ENDE         PIC X(03) VALUE LOW-VALUE.
	char 		END[3];
};

__global__ int comp3ToInt ( int memAddress, int length, int *currentRecordAttr ) {
	// converting BCD to integer 
	// http://www.3480-3590-data-conversion.com/article-bcd-binary.html
	return 0;
}

__global__ int comp3ToSignedInt ( int memAddress, int length, int *currentRecordAttr ) {
	return 0;
}

__global__ int charToCharArray ( int memAddress, int length ) {
	// converting ebcdic to ascii
	// https://stackoverflow.com/questions/7734275/c-code-to-convert-ebcdic-printables-to-ascii-in-place
	return 0;
}

__global__ char smallInt ( char in ) {
	return in;
}

__global__ void writeRecordToMemory ( int *currentRecord ) {
	return 0;
}

__global__ void decodeRecord ( int numOfCores, int recordNum ) {
	int inMemAddressBase;
	struct recordData theRecord;
	comp3ToInt ( inMemAddressBase, 5, &theRecord.RECID );
		inMemAddressBase = inMemAddressBase + 5; 
	charToCharArray ( inMemAddressBase, 4, &theRecord.SYSTEMID );
		inMemAddressBase = inMemAddressBase + 4; 
	comp3ToInt ( inMemAddressBase, 5, &theRecord.MANDID );
		inMemAddressBase = inMemAddressBase + 5; 
	charToCharArray ( inMemAddressBase, 40, &theRecord.NAME );
		inMemAddressBase = inMemAddressBase + 40; 
	comp3ToInt ( inMemAddressBase, 11, &theRecord.POLNR );
		inMemAddressBase = inMemAddressBase + 11; 
	comp3ToInt ( inMemAddressBase, 3, &theRecord.RISPA );
		inMemAddressBase = inMemAddressBase + 3; 
	charToCharArray ( inMemAddressBase, 3, &theRecord.WAEHR );
		inMemAddressBase = inMemAddressBase + 3; 
	comp3ToSignedInt ( inMemAddressBase, 9, &theRecord.PRAEMIE );
		inMemAddressBase = inMemAddressBase + 9; 
	charToCharArray ( inMemAddressBase, 50, &theRecord.INFO );
		inMemAddressBase = inMemAddressBase + 50; 
	charToCharArray ( inMemAddressBase, 3, &theRecord.ENDE );
		inMemAddressBase = inMemAddressBase + 3; 

}
