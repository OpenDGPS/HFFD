#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


// 03 C86R8200-RECID        PIC 9(05) COMP-3 VALUE ZERO.
__global__ int comp3ToInt ( int memAddress ) {
	// converting BCD to integer 
	// http://www.3480-3590-data-conversion.com/article-bcd-binary.html
	char byteOne, byteTwo, byteThree, byteFour, byteFive;
	byteOne = memAddress[0];
	byteTwo = memAddress[1];
	byteThree = memAddress[2];
	byteFour = memAddress[3];
	byteFive = memAddress[4];
	return 0;
}

__global__ char smallInt ( char in ) {
	return in;
}

__global__ void decodeRecord ( int numOfCores, int recordNum ) {
}
