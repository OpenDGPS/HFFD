#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

struct recordData
{
	uint32_t 	RECID;
	char		SYSTEMID[4];
	uint32_t	MANDID;
	char 		NAME[40];
};


// 03 RECID        PIC 9(05) COMP-3 VALUE ZERO.
// 03 MANDID       PIC 9(05) COMP-3 VALUE 10010.
// 03 POLNR        PIC 9(11) COMP-3 VALUE 0100001.
__global__ int comp3ToInt ( int memAddress, int length, int *currentRecordAttr ) {
	// converting BCD to integer 
	// http://www.3480-3590-data-conversion.com/article-bcd-binary.html
	char byteOne, byteTwo, byteThree, byteFour, byteFive;
	byteOne 	= memAddress[0];
	byteTwo 	= memAddress[1];
	byteThree 	= memAddress[2];
	byteFour 	= memAddress[3];
	byteFive 	= memAddress[4];
	return 0;
}

// 03 SYSTEMID     PIC X(04) VALUE 'S085'.
// 03 NAME         PIC X(40) VALUE 'MAX MUSTER'.
__global__ int charToCharArray ( int memAddress, int length ) {
	// converting ebcdic to ascii
	// https://stackoverflow.com/questions/7734275/c-code-to-convert-ebcdic-printables-to-ascii-in-place
	return 0;
}

__global__ char smallInt ( char in ) {
	return in;
}

__global__ void writeRecordToMemory ( int *currentRecord ) {
	return 0;
}

__global__ void decodeRecord ( int numOfCores, int recordNum ) {
	int inMemAddressBase;
	struct recordData theRecord;
	comp3ToInt ( inMemAddressBase, 5, &theRecord.RECID );
		inMemAddressBase = inMemAddressBase + 5; 
	charToCharArray ( inMemAddressBase, 4, &theRecord.SYSTEMID );
		inMemAddressBase = inMemAddressBase + 4; 
	comp3ToInt ( inMemAddressBase, 5, &theRecord.MANDID );
		inMemAddressBase = inMemAddressBase + 5; 
	charToCharArray ( inMemAddressBase, 40, &theRecord.NAME );
		inMemAddressBase = inMemAddressBase + 40; 
	comp3ToInt ( inMemAddressBase, 11, &theRecord.MANDID );
		inMemAddressBase = inMemAddressBase + 11; 
}
