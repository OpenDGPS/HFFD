#include "hip/hip_runtime.h"
/*****************************************************************/
/*	             THIS FILE WILL BE GENERATED                     */
/*	                      DO NOT EDIT!                           */
/*****************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "converter.cu"
#include "writeRecord.cu"

struct recordData
{
	// 03 RECID        PIC 9(05) COMP-3 VALUE ZERO.
	uint32_t 	RECID;
	// 03 SYSTEMID     PIC X(04) VALUE 'S085'.
	char		SYSTEMID[4];
	// 03 MANDID       PIC 9(05) COMP-3 VALUE 10010.
	uint32_t	MANDID;
	// 03 NAME         PIC X(40) VALUE 'MAX MUSTER'.
	char 		NAME[40];
	// 03 POLNR        PIC 9(11) COMP-3 VALUE 0100001.
	uint32_t 	POLNR;
	// 03 RISPA        PIC 9(03) COMP-3 VALUE 207.
	uint32_t 	RISPA;
	// 03 WAEHR        PIC X(03) VALUE 'EUR'.
	char 		WAEHR[3];
	// 03 PRAEMIE      PIC S9(9)V99 COMP-3 VALUE 228.30.
	int32_t		PRAEMIE;
	// 03 INFO         PIC X(50) VALUE 'ICH BIN EIN SATZ.'.
	char 		INFO[50];
	// 03 ENDE         PIC X(03) VALUE LOW-VALUE.
	char 		END[3];
};


__global__ void decodeRecord ( int numOfCores, int recordNum ) {
	int inMemAddressBase;
	struct recordData theRecord;
	comp3ToInt ( inMemAddressBase, 5, &theRecord.RECID );
		inMemAddressBase = inMemAddressBase + 5; 
	charToCharArray ( inMemAddressBase, 4, &theRecord.SYSTEMID );
		inMemAddressBase = inMemAddressBase + 4; 
	comp3ToInt ( inMemAddressBase, 5, &theRecord.MANDID );
		inMemAddressBase = inMemAddressBase + 5; 
	charToCharArray ( inMemAddressBase, 40, &theRecord.NAME );
		inMemAddressBase = inMemAddressBase + 40; 
	comp3ToInt ( inMemAddressBase, 11, &theRecord.POLNR );
		inMemAddressBase = inMemAddressBase + 11; 
	comp3ToInt ( inMemAddressBase, 3, &theRecord.RISPA );
		inMemAddressBase = inMemAddressBase + 3; 
	charToCharArray ( inMemAddressBase, 3, &theRecord.WAEHR );
		inMemAddressBase = inMemAddressBase + 3; 
	comp3ToSignedInt ( inMemAddressBase, 9, &theRecord.PRAEMIE );
		inMemAddressBase = inMemAddressBase + 9; 
	charToCharArray ( inMemAddressBase, 50, &theRecord.INFO );
		inMemAddressBase = inMemAddressBase + 50; 
	charToCharArray ( inMemAddressBase, 3, &theRecord.ENDE );
		inMemAddressBase = inMemAddressBase + 3; 

}
