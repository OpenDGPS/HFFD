
#include <hip/hip_runtime.h>

__device__ static const unsigned char e2a[256] = {
          0,  1,  2,  3,156,  9,134,127,151,141,142, 11, 12, 13, 14, 15,
         16, 17, 18, 19,157,133,  8,135, 24, 25,146,143, 28, 29, 30, 31,
        128,129,130,131,132, 10, 23, 27,136,137,138,139,140,  5,  6,  7,
        144,145, 22,147,148,149,150,  4,152,153,154,155, 20, 21,158, 26,
         32,160,161,162,163,164,165,166,167,168, 91, 46, 60, 40, 43, 33,
         38,169,170,171,172,173,174,175,176,177, 93, 36, 42, 41, 59, 94,
         45, 47,178,179,180,181,182,183,184,185,124, 44, 37, 95, 62, 63,
        186,187,188,189,190,191,192,193,194, 96, 58, 35, 64, 39, 61, 34,
        195, 97, 98, 99,100,101,102,103,104,105,196,197,198,199,200,201,
        202,106,107,108,109,110,111,112,113,114,203,204,205,206,207,208,
        209,126,115,116,117,118,119,120,121,122,210,211,212,213,214,215,
        216,217,218,219,220,221,222,223,224,225,226,227,228,229,230,231,
        123, 65, 66, 67, 68, 69, 70, 71, 72, 73,232,233,234,235,236,237,
        125, 74, 75, 76, 77, 78, 79, 80, 81, 82,238,239,240,241,242,243,
         92,159, 83, 84, 85, 86, 87, 88, 89, 90,244,245,246,247,248,249,
         48, 49, 50, 51, 52, 53, 54, 55, 56, 57,250,251,252,253,254,255
};

__device__ void comp3ToInt ( uint8_t *inputMemAddress, int fieldBaseAddress, int length, uint32_t *currentRecordAttr ) {
	// converting BCD to integer 
	// http://www.3480-3590-data-conversion.com/article-bcd-binary.html
	// return 0;
	currentRecordAttr[0] = inputMemAddress[fieldBaseAddress];
}

__device__ void comp3ToSignedInt ( int memAddress, int length, int *currentRecordAttr ) {
	// return 0;
}

__device__ void charToCharArray ( uint8_t *inputMemAddress, int fieldBaseAddress, int length, uint32_t *currentRecordAttr  ) {
	// converting ebcdic to ascii
	// https://stackoverflow.com/questions/7734275/c-code-to-convert-ebcdic-printables-to-ascii-in-place
	// return 0;

	currentRecordAttr[0] = e2a[inputMemAddress[fieldBaseAddress]];
}

__device__ void smallInt ( char in ) {
	// return in;
}
