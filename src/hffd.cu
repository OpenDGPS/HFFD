#include <stdio.h>
#include <stdlib.h>
#include "cuda/record.cu"


// 1317133


int getDeltaRecords ( char* inputMasterFileName, char* inputReferenceFileName, char* outputFileName ) {
  return 0;
}

int getDeltaVariables ( char* inputMasterFileName, char* inputReferenceFileName, char* outputFileName ) {
  return 0;
}

int main ( int argc, char **argv ) {
	printf("Starting kernel %d.\n", OUTPUTBINARYSIZE);
	uint8_t *ptr_hostfile;
	uint8_t *ptr_output;
	FILE *ptr_fp;
    uint8_t *d_hostfile;
    uint8_t *d_output;
	ptr_hostfile = (uint8_t *)malloc(BINARYSIZE);
	ptr_output = (uint8_t *)malloc(OUTPUTBINARYSIZE);
	if ( !ptr_hostfile ) {
		printf("Memory allocation error!\n");
		exit(1);
	} 
	if (argc < 2) {
		free(ptr_hostfile);
		free(ptr_output);
		printf("No input file.\n");
		exit(1);
	}
	if((ptr_fp = fopen(argv[1], "rb"))==NULL)
	{
		printf("Unable to open the file!\n");
		exit(1);
	}

	if(fread(ptr_hostfile, BINARYSIZE * sizeof( uint8_t ), 1, ptr_fp) != 1)
	{
		printf( "Read error!\n" );
		exit( 1 );
	}

	fclose(ptr_fp);
//	free(ptr_hostfile);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 blocksPerGrid(BLOCKSPERGRID,1,1); //use only one block
	dim3 threadsPerBlock(NUMOFCORES,1,1); //use N threads in the block myKernel<<<blocksPerGrid, threadsPerBlock>>>(result);
    
    checkCudaErrors(hipMalloc((uint8_t**)&d_hostfile, (BINARYSIZE)));
    checkCudaErrors(hipMemcpy(d_hostfile, ptr_hostfile, (BINARYSIZE), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((uint8_t**)&d_output, OUTPUTBINARYSIZE));
    checkCudaErrors(hipMemcpy(d_output, ptr_output, OUTPUTBINARYSIZE, hipMemcpyHostToDevice));
 	// start the i86 opcode interpreter on the GPU   
	hipEventRecord(start);
    decodeRecord<<<blocksPerGrid, threadsPerBlock>>>( d_hostfile, d_output ); // numofcores and recordnum
	hipEventRecord(stop);
    
    // checkCudaErrors(hipMemcpy(ptr_hostfileCopy, d_hostfile, (TOTALBINARYSIZE), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(ptr_output, d_output, OUTPUTBINARYSIZE, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_hostfile));
    checkCudaErrors(hipFree(d_output));
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("used time in sec: %f\n", milliseconds / 1000);
/*

	for ( int i = 0; i < (OUTPUTRECORDLENGTH * 10); i++ ) {
		if ( (i % OUTPUTRECORDLENGTH) == 0 ) printf("\n%d\t", i);
		printf("%02x ", ptr_output[i]);
	}
	printf("\n");
*/
	if (argc < 3) {
		free(ptr_output);
		printf("No output file.\n");
		exit(0);
	}

	FILE *out = fopen(argv[2], "wb");
	if ( out != NULL ) {
		const size_t wrote = fwrite(ptr_output, OUTPUTBINARYSIZE, 1, out);
		printf("Datei geschrieben: %lu\n", wrote);
	}

	
	free(ptr_output);
	printf("Kernel stopped.\n");
  return 0;
}
