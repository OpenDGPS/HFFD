#include <stdio.h>
#include <stdlib.h>
#include "cuda/record.cu"

#define BINARYSIZE 1560
#define NUMOFCORES 13


int decodeHF ( void ) {
	uint8_t *ptr_hostfile;
	uint8_t *ptr_output;
	FILE *ptr_fp;
    uint8_t *d_hostfile;
    uint8_t *d_output;
	ptr_hostfile = (uint8_t *)malloc(BINARYSIZE);
	ptr_output = (uint8_t *)malloc(BINARYSIZE);
	if ( !ptr_hostfile ) {
		printf("Memory allocation error!\n");
		exit(1);
	} 
	if((ptr_fp = fopen("sample/sample-hostfile.bin", "rb"))==NULL)
	{
		printf("Unable to open the file!\n");
		exit(1);
	}

	if(fread(ptr_hostfile, BINARYSIZE * sizeof( uint8_t ), 1, ptr_fp) != 1)
	{
		printf( "Read error!\n" );
		exit( 1 );
	}

	fclose(ptr_fp);
//	free(ptr_hostfile);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 blocksPerGrid(1,1,1); //use only one block
	dim3 threadsPerBlock(NUMOFCORES,1,1); //use N threads in the block myKernel<<<blocksPerGrid, threadsPerBlock>>>(result);
    
    checkCudaErrors(hipMalloc((uint8_t**)&d_hostfile, (BINARYSIZE)));
    checkCudaErrors(hipMemcpy(d_hostfile, ptr_hostfile, (BINARYSIZE), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((uint8_t**)&d_output, NUMOFCORES * BINARYSIZE));
    checkCudaErrors(hipMemcpy(d_output, ptr_output, NUMOFCORES * BINARYSIZE, hipMemcpyHostToDevice));
 	// start the i86 opcode interpreter on the GPU   
	hipEventRecord(start);
    decodeRecord<<<blocksPerGrid, threadsPerBlock>>>(1,1, d_hostfile, d_output); // numofcores and recordnum
	hipEventRecord(stop);
    
    // checkCudaErrors(hipMemcpy(ptr_hostfileCopy, d_hostfile, (TOTALBINARYSIZE), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(ptr_output, d_output, NUMOFCORES * BINARYSIZE, hipMemcpyDeviceToHost));

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);


	for ( int i = 0; i < 360; i++ ) {
		if ( (i % 120) == 0 ) printf("\n%d\t", i);
		printf("%02x ", ptr_output[i]);
	}
	printf("\n");

	printf("used time in sec: %f\n", milliseconds / 1000);
	
	// free(ptr_hostfile);
	return 0;
}

int getDeltaRecords ( char* inputMasterFileName, char* inputReferenceFileName, char* outputFileName ) {
  return 0;
}

int getDeltaVariables ( char* inputMasterFileName, char* inputReferenceFileName, char* outputFileName ) {
  return 0;
}

int main ( void ) {
	printf("Starting kernel.\n");
	decodeHF();
	printf("Kernel stopped.\n");
  return 0;
}
