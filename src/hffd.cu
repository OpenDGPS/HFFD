#include <stdio.h>
#include <stdlib.h>
#include "cuda/record.cu"

#define BINARYSIZE RECORDLENGTH * NUMOFRECORDS 
#define OUTPUTBINARYSIZE OUTPUTRECORDLENGTH * NUMOFRECORDS
#define NUMOFCORES 130


int decodeHF ( void ) {
	uint8_t *ptr_hostfile;
	uint8_t *ptr_output;
	FILE *ptr_fp;
    uint8_t *d_hostfile;
    uint8_t *d_output;
	ptr_hostfile = (uint8_t *)malloc(BINARYSIZE);
	ptr_output = (uint8_t *)malloc(OUTPUTBINARYSIZE);
	if ( !ptr_hostfile ) {
		printf("Memory allocation error!\n");
		exit(1);
	} 
	if((ptr_fp = fopen("sample/sample-hostfile-1000.bin", "rb"))==NULL)
	{
		printf("Unable to open the file!\n");
		exit(1);
	}

	if(fread(ptr_hostfile, BINARYSIZE * sizeof( uint8_t ), 1, ptr_fp) != 1)
	{
		printf( "Read error!\n" );
		exit( 1 );
	}

	fclose(ptr_fp);
//	free(ptr_hostfile);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	dim3 blocksPerGrid(100,1,1); //use only one block
	dim3 threadsPerBlock(NUMOFCORES,1,1); //use N threads in the block myKernel<<<blocksPerGrid, threadsPerBlock>>>(result);
    
    checkCudaErrors(hipMalloc((uint8_t**)&d_hostfile, (BINARYSIZE)));
    checkCudaErrors(hipMemcpy(d_hostfile, ptr_hostfile, (BINARYSIZE), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((uint8_t**)&d_output, OUTPUTBINARYSIZE));
    checkCudaErrors(hipMemcpy(d_output, ptr_output, OUTPUTBINARYSIZE, hipMemcpyHostToDevice));
 	// start the i86 opcode interpreter on the GPU   
	hipEventRecord(start);
    decodeRecord<<<blocksPerGrid, threadsPerBlock>>>(1,1, d_hostfile, d_output); // numofcores and recordnum
	hipEventRecord(stop);
    
    // checkCudaErrors(hipMemcpy(ptr_hostfileCopy, d_hostfile, (TOTALBINARYSIZE), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(ptr_output, d_output, OUTPUTBINARYSIZE, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_hostfile));
    checkCudaErrors(hipFree(d_output));
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);


	// for ( int i = (1560000 - 1560); i < 1560000; i++ ) {
	for ( int i = 0; i < 1820; i++ ) {
		if ( (i % 140) == 0 ) printf("\n%d\t", i);
		printf("%02x ", ptr_output[i]);
	}
	printf("\n");

	FILE *out = fopen("../result.bin", "wb");
	if ( out != NULL ) {
		const size_t wrote = fwrite(ptr_output, OUTPUTBINARYSIZE, 1, out);
		printf("Datei geschrieben: %lu\n", wrote);
	}

	printf("used time in sec: %f\n", milliseconds / 1000);
	
	free(ptr_output);
	return 0;
}

int getDeltaRecords ( char* inputMasterFileName, char* inputReferenceFileName, char* outputFileName ) {
  return 0;
}

int getDeltaVariables ( char* inputMasterFileName, char* inputReferenceFileName, char* outputFileName ) {
  return 0;
}

int main ( void ) {
	printf("Starting kernel %d.\n", OUTPUTBINARYSIZE);
	decodeHF();
	printf("Kernel stopped.\n");
  return 0;
}
